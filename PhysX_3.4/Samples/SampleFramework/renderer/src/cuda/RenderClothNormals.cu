#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2018 NVIDIA Corporation. All rights reserved.

#include "PxPhysics.h"
#include "PxVec4.h"
#include "PxVec3.h"
#include "PxVec2.h"

#include "cloth/PxClothTypes.h"

namespace physx
{

// interleaved format must match that used by RendererClothShape
struct Vertex
{
	PxVec3 position;
	PxVec3 normal;
};

namespace
{
	__device__ inline void PxAtomicFloatAdd(float* dest, float x)
	{
#if __CUDA_ARCH__ >= 200
		atomicAdd(dest, x);
#else
		union bits { float f; unsigned int i; };
		bits oldVal, newVal;

		do
		{
			// emulate atomic float add on 1.1 arch
			oldVal.f = *dest;
			newVal.f = oldVal.f + x;
		}
		while (atomicCAS((unsigned int*)dest, oldVal.i, newVal.i) != oldVal.i);
#endif
	}


	__device__ void PxAtomicVec3Add(PxVec3& dest, PxVec3 inc)
	{
		PxAtomicFloatAdd(&dest.x, inc.x);
		PxAtomicFloatAdd(&dest.y, inc.y);
		PxAtomicFloatAdd(&dest.z, inc.z);
	}
}

extern "C" __global__ void computeSmoothNormals(
	const PxClothParticle* particles,
	const PxU16* indices,
	Vertex* vertices,
	PxU32 numTris,
	PxU32 numParticles)
{
	// zero old normals
	for (PxU32 i=threadIdx.x; i < numParticles; i += blockDim.x)
		vertices[i].normal = PxVec3(0.0f);

	__syncthreads();

	for (PxU32 i=threadIdx.x; i < numTris; i += blockDim.x)
	{
		PxU16 a = indices[i*3];
		PxU16 b = indices[i*3+1];
		PxU16 c = indices[i*3+2];

		// calculate face normal
		PxVec3 e1 = particles[b].pos-particles[a].pos;
		PxVec3 e2 = particles[c].pos-particles[a].pos;
		PxVec3 n = e2.cross(e1);

		PxAtomicVec3Add(vertices[a].normal, n);
		PxAtomicVec3Add(vertices[b].normal, n);
		PxAtomicVec3Add(vertices[c].normal, n);
	}

	__syncthreads();

	// update vertex buffer
	for (PxU32 i=threadIdx.x; i < numParticles; i += blockDim.x)
	{
		vertices[i].position = particles[i].pos;
		vertices[i].normal = vertices[i].normal.getNormalized();
	}	
}

}