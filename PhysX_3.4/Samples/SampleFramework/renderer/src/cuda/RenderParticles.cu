#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2018 NVIDIA Corporation. All rights reserved.

#include "PxPhysics.h"
#include "PxVec4.h"
#include "PxVec3.h"
#include "PxVec2.h"
#include "PxMat33.h"
#include "PxStrideIterator.h"

namespace physx
{

template <typename T>
__device__ T* ptrOffset(T* p, PxU32 byteOffset)
{
	return (T*)((unsigned char*)(p) + byteOffset);
}

#if __CUDA_ARCH__ < 200
__device__ PxU32 gOffset;
#else
__device__ __shared__ PxU32 gOffset;
#endif


// copies orientations and positions to the destination vertex
// buffer based on the validityBitmap state
extern "C" __global__ void updateInstancedVB(
	PxVec3* destPositions,
	PxVec3* destRotation0,
	PxVec3* destRotation1,
	PxVec3* destRotation2,
	PxU32 destStride,
	const PxVec4* srcPositions,
	const PxMat33* srcRotations,
	const PxU32* validParticleBitmap,
	PxU32 validParticleRange)
{
	if (!threadIdx.x)
		gOffset = 0;

	__syncthreads();

	if (validParticleRange)
	{
		for (PxU32 w=threadIdx.x; w <= (validParticleRange) >> 5; w+=blockDim.x)
		{
			const PxU32 srcBaseIndex = w << 5;

			// reserve space in the output vertex buffer based on
			// population count of validity bitmap (avoids excess atomic ops)
			PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));
			
			for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
			{
				const PxU32 index = srcBaseIndex | __ffs(b)-1;

				const PxU32 offset = destIndex*destStride;

				*ptrOffset(destRotation0, offset) = srcRotations[index].column0;
				*ptrOffset(destRotation1, offset) = srcRotations[index].column1;
				*ptrOffset(destRotation2, offset) = srcRotations[index].column2;

				PxVec3* p = ptrOffset(destPositions, offset);
				p->x = srcPositions[index].x;
				p->y = srcPositions[index].y;
				p->z = srcPositions[index].z;

				++destIndex;
			}
		}
	}
}


// copies positions and alpha to the destination vertex buffer based on 
// validity bitmap and particle life times
extern "C" __global__ void updateBillboardVB(
	PxVec3* destPositions,
	PxU8* destAlphas,
	PxU32 destStride,
	PxF32 fadingPeriod,
	const PxVec4* srcPositions, 
	const PxReal* srcLifetimes,
	const PxU32* validParticleBitmap,
	PxU32 validParticleRange)
{
	if (!threadIdx.x)
		gOffset = 0;

	__syncthreads();

	if (validParticleRange)
	{
		for (PxU32 w=threadIdx.x; w <= (validParticleRange) >> 5; w+=blockDim.x)
		{
			const PxU32 srcBaseIndex = w << 5;

			// reserve space in the output vertex buffer based on
			// population count of validity bitmap (avoids excess atomic ops)
			PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));

			for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
			{
				PxU32 index = srcBaseIndex | __ffs(b)-1;

				const PxU32 offset = destIndex*destStride;

				// copy position
				PxVec3* p = ptrOffset(destPositions, offset);
				p->x = srcPositions[index].x;
				p->y = srcPositions[index].y;
				p->z = srcPositions[index].z;

				// update alpha
				if (srcLifetimes)
				{
					PxU8 lifetime = 0;
					if(srcLifetimes[index] >= fadingPeriod)
						lifetime = 255;
					else
					{
						if(srcLifetimes[index] <= 0.0f)
							lifetime = 0; 
						else
							lifetime = static_cast<PxU8>(srcLifetimes[index] * 255 / fadingPeriod);
					}

					destAlphas[destIndex*4] = lifetime;
				}

				++destIndex;
			}
		}
	}
}

}